
#include <hip/hip_runtime.h>
#include <stdio.h>
/* Host = CPU stuff
   Device = GPU Stuff
 */

__global__ void mykernel(void){
  /*global indicates a function that runs on device called from
    host */
} 


int main(void)
{
  mykernel<<<1,1>>>(); //Calls from host to device
  printf("hello world! \n");
  return 0;
}
